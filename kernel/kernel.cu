#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "math.h"
#include "io.hpp"

__global__ void step(int *arr, int *result, size_t N, int width) {
    int index =  blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

   for(int i = index; i < N; i += stride)
   {
        int live_neighbours = 0;
        int neighbour_indexes[8];

        neighbour_indexes[0] = (i - width) - 1; // top left
        neighbour_indexes[1] = (i - width); // top
        neighbour_indexes[2] = (i - width) + 1; // top right

        neighbour_indexes[3] = (i - 1); // left
        neighbour_indexes[4] = (i + 1); // right

        neighbour_indexes[5] = (i + width) - 1; // bottom left
        neighbour_indexes[6] = (i + width); // bottom
        neighbour_indexes[7] = (i + width) + 1; // bottom right


        // if the top left isn't at the end of the line or before the array
        if (!(neighbour_indexes[0] < 0 || neighbour_indexes[0] % width == (width - 1))) {
            if (arr[neighbour_indexes[0]]) {
                live_neighbours++;
            }
        }

        // if the top one does exist 
        if (!(neighbour_indexes[1] < 0)) {
            if (arr[neighbour_indexes[1]]) {
                live_neighbours++;
            }
        }

        // if the top right isn't at the start of a line or before the array
        if (!(neighbour_indexes[2] < 0 || neighbour_indexes[2] % width == 0)) {
            if (arr[neighbour_indexes[2]]) {
                live_neighbours++;
            }
        }

        // if the left isn't at the end of a line
        if (!(neighbour_indexes[3] % width == (width - 1)) || neighbour_indexes[3] < 0) {
            if (arr[neighbour_indexes[3]]) {
                live_neighbours++;
            }
        }

        // if the right isn't at the start of the next line
        if (!(neighbour_indexes[4] % width == 0) || neighbour_indexes[4] > N) {
            if (arr[neighbour_indexes[4]]) {
                live_neighbours++;
            }
        }

        // if the bottom left isn't at the end of a line
        if (!(neighbour_indexes[5] > N || neighbour_indexes[5] % width == (width - 1))) {
            if (arr[neighbour_indexes[5]]) {
                live_neighbours++;
            }
        }

        // if the bottom one isn't out of the array
        if (neighbour_indexes[6] < N) {
            if (arr[neighbour_indexes[6]]) {
                live_neighbours++;
            }
        }

        // if the bottom right isn't at the start of a line or out of the array
        if (!(neighbour_indexes[7] > N || neighbour_indexes[7] % width == 0)) {
            if (arr[neighbour_indexes[7]]) {
                live_neighbours++;
            }
        }

        // -----------------------------------------
        

        //printf("Cell %d has %d ln \n" , i , live_neighbours);

        if (arr[i] && (live_neighbours == 2 || live_neighbours == 3)) {
            result[i] = 1;
        }else if (!arr[i] && (live_neighbours == 3))
        {
            result[i] = 1;
        }else {
            if(arr[i]) {
                result[i] = 0;
            }else if (!arr[i]) {
                result[i] = 0;
            }
        }
        

   }
}



hipDeviceProp_t getDetails(int deviceId)
{
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, deviceId);
    return props;
}



inline __global__ void copy(int *result, int *input, int N) {
    int index =  blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < N; i += stride) {
        input[i] = result[i]; 
    }
}

#define multi 20
void cgol::conways_game_of_life(int *input, int N, int generations, bool save) {

    printf("Initialising GPU...\n");
    int deviceId;
    hipGetDevice(&deviceId);
    hipDeviceProp_t props = getDetails(deviceId);


    size_t size = sizeof(int) * N;

    int *result;
    int *d_result;
    int *d_input;

    result = (int*)malloc(size);

    hipMalloc((void **)&d_input, size);
    hipMalloc((void**)&d_result, size);
    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);
    
    int threads_per_block = 512;
    printf("Number of SMs : %d\n\r", props.multiProcessorCount);
    int number_of_blocks = props.multiProcessorCount * multi;

    hipError_t step_error;
    hipError_t async_error;

    int width = (int)sqrt(N); // Because it's a square this should always be true.

    if (width * width != N) {
        // check to make sure it is
        printf("Err: width is not correct. \n %d * %d != %d", width, width, N);
        exit(-2);
    }

    printf("Executing:\n");

    for (int i = 0; i < generations; i++) {
        step<<<threads_per_block, number_of_blocks>>>(d_input, d_result, N, width);
        
        // check for errors
        step_error = hipGetLastError();
        if(step_error != hipSuccess) printf("Error: %s\n", hipGetErrorString(step_error));

        async_error = hipDeviceSynchronize();
        if(async_error != hipSuccess) printf("Error: %s\n", hipGetErrorString(async_error));

        // result now contains the result of cgol.
        // we want to write that to a file and then copy that to d_input and run again.
        if(save) {
            hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost);
            cgol::write("./out/map" + std::to_string(i) + ".mp", result, N);
        }
        
        // copy result as d_input ready for the next kernel call
        copy<<<threads_per_block,number_of_blocks>>>(d_result, d_input, N);
    }

}